#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "..\usr\include\GL\freeglut.h"
#include <vector>
#include <time.h>
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include "thrust/copy.h"

#define NumParticle 1000
#define TILE_WIDTH 32

using namespace std;

int Width = 800;
int Height = 600;

struct Particle {
	double m;		// 질량
	double x[3];	// 위치
	double v[3];	// 속도
	double size;	// 크기
	double c[3];	// 색상
	double age;		// 나이

	bool launch;		// 발사 여부
	double launchTime;	// 발사 시간
	double launchV[3];	// 발사 시간 동안의 속도
};

double downGravity[3] = { 0.0, -9.8, 0.0 };
double upGravity[3] = { 0.0, -1.2, 0.0 };

vector<Particle> PSystem;

void Render();
void Reshape(int w, int h);
void Mouse(int button, int state, int x, int y);
void Keyboard(unsigned char key, int x, int y);
void Timer(int id);

void iter(double dt, vector<Particle>::iterator it);

__global__ void myKernel(Particle* ptr, double dt, int width, thrust::device_vector<Particle> DSystem);

int main(int argc, char** argv) {
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);

	glutInitWindowSize(Width, Height);
	glutCreateWindow("Firework System GPU");

	glutDisplayFunc(Render);
	glutReshapeFunc(Reshape);
	glutMouseFunc(Mouse);
	glutKeyboardFunc(Keyboard);
	glutTimerFunc(10, Timer, 0);

	glutMainLoop();
	return 0;
}

void Reshape(int w, int h) {
	glViewport(0, 0, w, h);
}

__global__ void myKernel(Particle* ptr, double dt, int width, thrust::device_vector<Particle> DSystem) {
	int i = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int j = blockIdx.x * TILE_WIDTH + threadIdx.x;

	if (i < width && j < width) {
		if (ptr[i * width + j].launch) {
			ptr[i * width + j].x[0] = ptr[i * width + j].x[0] + dt * ptr[i * width + j].launchV[0];
			ptr[i * width + j].x[1] = ptr[i * width + j].x[1] + dt * ptr[i * width + j].launchV[1];
			ptr[i * width + j].x[2] = ptr[i * width + j].x[2] + dt * ptr[i * width + j].launchV[2];

			ptr[i * width + j].launchV[1] = ptr[i * width + j].launchV[1] + dt * (-1.2);

			if (ptr[i * width + j].launchV[1] < 0.0f) {
				ptr[i * width + j].launch = false;
			}
		}
		else {
			ptr[i * width + j].x[0] = ptr[i * width + j].x[0] + dt * ptr[i * width + j].v[0];
			ptr[i * width + j].x[1] = ptr[i * width + j].x[1] + dt * ptr[i * width + j].v[1];
			ptr[i * width + j].x[2] = ptr[i * width + j].x[2] + dt * ptr[i * width + j].v[2];

			ptr[i * width + j].v[1] = ptr[i * width + j].v[1] + dt * (-9.8 / ptr[i * width + j].m);
		}

		ptr[i * width + j].age -= 0.1;
	}
}

void iter(double dt, vector<Particle>::iterator it) {
	thrust::device_vector<Particle> DSystem = PSystem;

	if (!PSystem.empty()) {
		int size = PSystem.size();
		Particle* raw_ptr = thrust::raw_pointer_cast(&DSystem[0]);

		myKernel << <((size - 1) / TILE_WIDTH + 1, (size - 1) / TILE_WIDTH + 1), (TILE_WIDTH, TILE_WIDTH) >> > (raw_ptr, dt, PSystem.size(), DSystem);

		thrust::copy(DSystem.begin(), DSystem.end(), PSystem.begin());
	}

	while (it != PSystem.end()) {
		int count = 0;

		if (it->age < 0.0) {
			it = PSystem.erase(it);
			continue;
		}

		if (it->m > 19.9 && (it->age < 0.3 && it->age > 0.2)) {
			double x0 = it->x[0];
			double x1 = it->x[1];

			double c[3] = { it->c[0], it->c[1], it->c[2] };
			for (int i = 0; i < 500; i++) {
				Particle p;
				p.m = rand() / (double)RAND_MAX * 10.0;

				p.x[0] = x0;
				p.x[1] = x1;
				p.x[2] = 0.0;

				double theta = 2 * 3.14 * (double)i / (500 - 1);
				double speed = rand() / (double)RAND_MAX * 10.0f;
				p.v[0] = speed * cos(theta);
				p.v[1] = speed * sin(theta);
				p.v[2] = 0.0;

				p.size = rand() / (double)RAND_MAX * 3.5;

				p.age = p.m;
				p.c[0] = c[0];
				p.c[1] = c[1];
				p.c[2] = c[2];

				p.launch = false;
				p.launchTime = 0.0f;
				p.age = p.m + p.launchTime;

				PSystem.push_back(p);
			}
		}

		count++;
		it = PSystem.begin() + count;
	}
}

void Timer(int id) {
	clock_t st = clock();
	double dt = 0.1;

	//thrust::device_vector<Particle> DSystem = PSystem;
	vector<Particle>::iterator it = PSystem.begin();

	iter(dt, it);

	glutPostRedisplay();
	//printf("particle count = %d\n", PSystem.size());
	//printf("Elapsed time = %u ms\n", clock() - st);
	glutTimerFunc(10, Timer, 0);
}

void Mouse(int button, int state, int x, int y) {
	int randParticle = 700 + int(rand() / (double)RAND_MAX * NumParticle); // 700 < randParticle < 1700

	double r = 0.5 + rand() / ((double)RAND_MAX * 2); // 밝은 색을 내기 위하여 색 값은 0.5 ~ 1.0 사이로 설정
	double g = 0.5 + rand() / ((double)RAND_MAX * 2);
	double b = 0.5 + rand() / ((double)RAND_MAX * 2);
	double time = randParticle / (double)100.0f;	// 수명은 생성되는 파티클 수에 비례

	double launchSpeed = 15 + (double)randParticle / 200;	// 발사 속도 역시 생성되는 파티클 수에 비례

	for (int i = 0; i < randParticle; ++i) {
		Particle p;

		p.m = rand() / (double)RAND_MAX * 20.0;

		p.x[0] = x;
		p.x[1] = Height - y;
		p.x[2] = 0.0;

		double theta = 2 * 3.14 * (double)i / (randParticle - 1);
		double speed = rand() / (double)RAND_MAX * 10.0f;
		p.v[0] = speed * cos(theta);
		p.v[1] = speed * sin(theta);
		p.v[2] = 0.0;

		p.launchV[0] = 0.0f;
		p.launchV[1] = launchSpeed;
		p.launchV[2] = 0.0f;

		p.size = rand() / (double)RAND_MAX * 5.0;

		p.c[0] = r;
		p.c[1] = g;
		p.c[2] = b;

		p.launch = true;
		p.launchTime = time;
		p.age = p.m + p.launchTime * 1.2;

		PSystem.push_back(p);
	}
}

void Render() {
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, Width, 0.0, Height, -10.0, 10.0);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	glEnable(GL_POINT_SMOOTH);
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

	for (int i = 0; i < PSystem.size(); ++i) {
		glPointSize(PSystem[i].size);
		glBegin(GL_POINTS);
		glColor3dv(PSystem[i].c);
		glVertex3dv(PSystem[i].x);
		glEnd();
	}

	glutSwapBuffers();
}

void Keyboard(unsigned char key, int x, int y) {
	if (key == 27) exit(1);
}