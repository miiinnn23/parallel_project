#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "..\usr\include\GL\freeglut.h"
#include <vector>
#include <time.h>
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include "thrust/copy.h"

#define NumParticle 1000
#define TILE_WIDTH 32

using namespace std;

int Width = 800;
int Height = 600;

struct Particle {
	double m;		// 질량
	double x[3];	// 위치
	double v[3];	// 속도
	double size;	// 크기
	double c[3];	// 색상
	double age;		// 나이
};

double Gravity[3] = { 0.0, -9.8, 0.0 };
double ExtForce[3] = { 0.0, 0.0, 0.0 };
vector<Particle> PSystem;

void Render();
void Reshape(int w, int h);
void Mouse(int button, int state, int x, int y);
void Keyboard(unsigned char key, int x, int y);
void Timer(int id);

void iter(double dt, vector<Particle>::iterator it);

__global__ void myKernel(Particle* ptr, double dt, int width);

int main(int argc, char** argv) {
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);

	glutInitWindowSize(Width, Height);
	glutCreateWindow("Particle System");

	glutDisplayFunc(Render);
	glutReshapeFunc(Reshape);
	glutMouseFunc(Mouse);
	glutKeyboardFunc(Keyboard);
	glutTimerFunc(10, Timer, 0);

	glutMainLoop();
	return 0;
}

void Reshape(int w, int h) {
	glViewport(0, 0, w, h);
}

__global__ void myKernel(Particle* ptr, double dt, int width) {
	int i = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int j = blockIdx.x * TILE_WIDTH + threadIdx.x;
	//int i = blockDim.x * blockIdx.x + threadIdx.x;

	//if (i < width) {
	//	ptr[i].x[0] = ptr[i].x[0] + dt * ptr[i].v[0];
	//	ptr[i].x[1] = ptr[i].x[1] + dt * ptr[i].v[1];
	//	ptr[i].x[2] = ptr[i].x[2] + dt * ptr[i].v[2];

	//	ptr[i].v[1] = ptr[i].v[1] + dt * (-9.8 / ptr[i].m);
	//	ptr[i].age -= 0.1;
	//	//if(i == 0) printf("ptr[%d].age = %f\n", i, ptr[i].age);
	//}
	if (i < width && j < width) {
		/*if (i == 0 && j == 0) {
			printf("ptr[%d].age = %f\n", i * width + j, ptr[i * width + j].age);
		}*/
		ptr[i * width + j].x[0] = ptr[i * width + j].x[0] + dt * ptr[i * width + j].v[0];
		ptr[i * width + j].x[1] = ptr[i * width + j].x[1] + dt * ptr[i * width + j].v[1];
		ptr[i * width + j].x[2] = ptr[i * width + j].x[2] + dt * ptr[i * width + j].v[2];

		ptr[i * width + j].v[1] = ptr[i * width + j].v[1] + dt * (-9.8 / ptr[i * width + j].m);

		ptr[i * width + j].age -= 0.1;
	}
}

void iter(double dt, vector<Particle>::iterator it) {
	int count = 0;
	thrust::device_vector<Particle> DSystem = PSystem;

	if (!PSystem.empty()) {
		dim3 dimGrid((PSystem.size() - 1) / TILE_WIDTH + 1, (PSystem.size() - 1) / TILE_WIDTH + 1);
		dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

		Particle* raw_ptr = thrust::raw_pointer_cast(&DSystem[0]);

		//myKernel << <1, PSystem.size() >> > (raw_ptr, dt, PSystem.size());
		myKernel << <dimGrid, dimBlock>> > (raw_ptr, dt, PSystem.size());

		thrust::copy(DSystem.begin(), DSystem.end(), PSystem.begin());
	}

	while (it != PSystem.end()) {
		if (it->age < 0.0) {
			it = PSystem.erase(it);
			continue;
		}

		count++;
		it = PSystem.begin() + count;
	}
}

void Timer(int id) {
	clock_t st = clock();
	double dt = 0.1;

	//thrust::device_vector<Particle> DSystem = PSystem;
	vector<Particle>::iterator it = PSystem.begin();

	iter(dt, it);

	glutPostRedisplay();
	printf("particle count = %d\n", PSystem.size());
	//printf("Elapsed time = %u ms\n", clock() - st);
	glutTimerFunc(10, Timer, 0);
}

void Mouse(int button, int state, int x, int y) {
	int randParticle = int(rand() / (double)RAND_MAX * NumParticle);

	double r = rand() / (double)RAND_MAX * 1.4;
	double g = rand() / (double)RAND_MAX * 1.4;
	double b = rand() / (double)RAND_MAX * 1.4;
	for (int i = 0; i < randParticle; ++i) {
		Particle p;

		p.m = rand() / (double)RAND_MAX * 20.0;

		p.x[0] = x;
		p.x[1] = Height - y;
		p.x[2] = 0.0;

		double theta = 2 * 3.14 * (double)i / (randParticle - 1);
		double speed = rand() / (double)RAND_MAX * 10.0f;
		p.v[0] = speed * cos(theta);
		p.v[1] = speed * sin(theta);
		p.v[2] = 0.0;

		p.size = rand() / (double)RAND_MAX * 3.5;

		p.age = p.m * 0.8;
		p.c[0] = r;
		p.c[1] = g;
		p.c[2] = b;

		PSystem.push_back(p);
	}
}

void Render() {
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, Width, 0.0, Height, -10.0, 10.0);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	glEnable(GL_POINT_SMOOTH);
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

	for (int i = 0; i < PSystem.size(); ++i) {
		glPointSize(PSystem[i].size);
		glBegin(GL_POINTS);
		glColor3dv(PSystem[i].c);
		glVertex3dv(PSystem[i].x);
		glEnd();
	}

	glutSwapBuffers();
}

void Keyboard(unsigned char key, int x, int y) {
	if (key == 27) exit(1);

	if (key == '1') {
		ExtForce[0] = 100.0;
		ExtForce[1] = 0.0;
		ExtForce[2] = 0.0;
	}
	if (key == '2') {
		ExtForce[0] = -100.0;
		ExtForce[1] = 0.0;
		ExtForce[2] = 0.0;
	}
}