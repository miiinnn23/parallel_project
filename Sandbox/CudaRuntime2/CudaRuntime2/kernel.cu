#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "..\usr\include\GL\glew.h"
#include "..\usr\include\GL\freeglut.h"
#include "cuda_gl_interop.h"
#include <vector>
#include <time.h>
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include "thrust/copy.h"
#include <chrono>

#define NumParticle 1000
#define TILE_WIDTH 32

using namespace std;

int Width = 800;
int Height = 600;
GLuint gl_pbo;					// 픽셀 버퍼를 가르키는 OpenGL 핸들 
hipGraphicsResource* cuda_pbo;	// 픽셀 버퍼를 가르키는 CUDA 핸들 
uchar4* pDevImage;

struct Particle {
	double m;		// 질량
	double x[3];	// 위치
	double v[3];	// 속도
	double size;	// 크기
	double c[3];	// 색상
	double age;		// 나이

	bool launch;		// 발사 여부
	double launchTime;	// 발사 시간
	double launchV[3];	// 발사 시간 동안의 속도
};

int mouseCount = 0;
long timeMax = 0;

double downGravity[3] = { 0.0, -9.8, 0.0 };
double upGravity[3] = { 0.0, -1.2, 0.0 };

double ExtForce[3] = { 0.0, 0.0, 0.0 };
vector<Particle> PSystem;

void Render();
void Reshape(int w, int h);
void Mouse(int button, int state, int x, int y);
void Keyboard(unsigned char key, int x, int y);
void Timer(int id);
void Close();

void iter(double dt, vector<Particle>::iterator it);

__global__ void myKernel(Particle* ptr, double dt, int width, thrust::device_vector<Particle> DSystem);
//__global__ void beforePushBackKernel(Particle* ptr, double )

int main(int argc, char** argv) {
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);

	glutInitWindowSize(Width, Height);
	glutCreateWindow("Particle System");

	glutDisplayFunc(Render);
	glutReshapeFunc(Reshape);
	glutMouseFunc(Mouse);
	glutKeyboardFunc(Keyboard);
	glutTimerFunc(10, Timer, 0);
	glutCloseFunc(Close);

	glewInit();

	// OpenGL과 연동할 GPU 선택
	hipSetDevice(3);
	cudaGLSetGLDevice(3);

	// 1. 픽셀 버퍼에 대한 핸들을 생성하여 버퍼를 바인딩
	glGenBuffers(1, &gl_pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pbo);

	// 2. 픽셀 버퍼 할당
	//	파라미터 설명:
	//		NULL: 초기화 데이터 없음
	//		GL_DYNAMIC_DRAW_ARB: 버퍼가 반복적으로 수정될 것임	
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, Height * Width * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW_ARB);


	glutMainLoop();
	return 0;
}

void Reshape(int w, int h) {
	glViewport(0, 0, w, h);
}

void Close() {
	hipDeviceReset();

	// 픽셀 버퍼의 바인딩을 해제하고 핸들을 제거한다.
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
	glDeleteBuffers(1, &gl_pbo);
}
__global__ void myKernel(Particle* ptr, double dt, int width, thrust::device_vector<Particle> DSystem) {
	int i = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int j = blockIdx.x * TILE_WIDTH + threadIdx.x;

	if (i < width && j < width) {
		if (ptr[i * width + j].launch) {
			ptr[i * width + j].x[0] = ptr[i * width + j].x[0] + dt * ptr[i * width + j].launchV[0];
			ptr[i * width + j].x[1] = ptr[i * width + j].x[1] + dt * ptr[i * width + j].launchV[1];
			ptr[i * width + j].x[2] = ptr[i * width + j].x[2] + dt * ptr[i * width + j].launchV[2];

			ptr[i * width + j].launchV[1] = ptr[i * width + j].launchV[1] + dt * (-1.2);

			if (ptr[i * width + j].launchV[1] < 0.0f) {
				ptr[i * width + j].launch = false;
			}
		}
		else {
			ptr[i * width + j].x[0] = ptr[i * width + j].x[0] + dt * ptr[i * width + j].v[0];
			ptr[i * width + j].x[1] = ptr[i * width + j].x[1] + dt * ptr[i * width + j].v[1];
			ptr[i * width + j].x[2] = ptr[i * width + j].x[2] + dt * ptr[i * width + j].v[2];

			ptr[i * width + j].v[1] = ptr[i * width + j].v[1] + dt * (-9.8 / ptr[i * width + j].m);
		}

		ptr[i * width + j].age -= 0.1;
	}
}

__global__ void beforePushBackKernel(Particle* ptr, double) {

}

void iter(double dt, vector<Particle>::iterator it) {
	int count = 0;
	thrust::device_vector<Particle> DSystem = PSystem;

	if (!PSystem.empty()) {
		int size = PSystem.size();
		Particle* raw_ptr = thrust::raw_pointer_cast(&DSystem[0]);

		myKernel << <((size - 1) / TILE_WIDTH + 1, (size - 1) / TILE_WIDTH + 1), (TILE_WIDTH, TILE_WIDTH) >> > (raw_ptr, dt, PSystem.size(), DSystem);

		thrust::copy(DSystem.begin(), DSystem.end(), PSystem.begin());
	}

	while (it != PSystem.end()) {
		if (it->age < 0.0) {
			it = PSystem.erase(it);
			continue;
		}
		if (it->m > 19.9 && (it->age < 0.3 && it->age > 0.2)) {
			double x0 = it->x[0];
			double x1 = it->x[1];


			double c[3] = { it->c[0], it->c[1], it->c[2] };

			for (int i = 0; i < 500; i++) {
				Particle p;
				p.m = rand() / (double)RAND_MAX * 10.0;

				p.x[0] = x0;
				p.x[1] = x1;
				p.x[2] = 0.0;

				double theta = 2 * 3.14 * (double)i / (500 - 1);
				double speed = rand() / (double)RAND_MAX * 10.0f;
				p.v[0] = speed * cos(theta);
				p.v[1] = speed * sin(theta);
				p.v[2] = 0.0;

				p.size = rand() / (double)RAND_MAX * 3.5;

				p.age = p.m;
				p.c[0] = c[0];
				p.c[1] = c[1];
				p.c[2] = c[2];

				p.launch = false;
				p.launchTime = 0.0f;
				p.age = p.m + p.launchTime;
			/*Particle* ptr, dPtr;
			hipMalloc((Particle**)&dPtr, sizeof(Particle) * 500);
			hipMemcpy(dPtr, ptr, sizeof(Particle) * 500, hipMemcpyHostToDevice);
			beforePushBackKernel << < >> > (Particle * ptr, double);*/

				PSystem.push_back(p);
			}
			
		}
		count++;
		it = PSystem.begin() + count;
	}
}

void Timer(int id) {
	clock_t st = clock();
	double dt = 0.1;

	//thrust::device_vector<Particle> DSystem = PSystem;
	vector<Particle>::iterator it = PSystem.begin();

	iter(dt, it);

	glutPostRedisplay();

	long time = clock() - st;
	//timeMax = (time > timeMax) ? time : timeMax;
	if (mouseCount != 1) {
		timeMax = (time > timeMax) ? time : timeMax;
	}

	//printf("clicked %d times : Elapsed time = %u ms\n", mouseCount, time);
	if (mouseCount != 0 && PSystem.empty()) {
		printf("clicked %d times : max time = %u ms\n", mouseCount, timeMax);
		timeMax = 0;
		mouseCount = 0;
	}
	//printf("particle count = %d\n", PSystem.size());
	//printf("Elapsed time = %u ms\n", clock() - st);
	glutTimerFunc(10, Timer, 0);
}

void Mouse(int button, int state, int x, int y) {
	mouseCount++;
	int randParticle = 700 + int(rand() / (double)RAND_MAX * NumParticle); // 700 < randParticle < 1700

	double r = 0.5 + rand() / ((double)RAND_MAX * 2); // 밝은 색을 내기 위하여 색 값은 0.5 ~ 1.0 사이로 설정
	double g = 0.5 + rand() / ((double)RAND_MAX * 2);
	double b = 0.5 + rand() / ((double)RAND_MAX * 2);
	double time = randParticle / (double)100.0f;	// 수명은 생성되는 파티클 수에 비례

	double launchSpeed = 15 + (double)randParticle / 200;	// 발사 속도 역시 생성되는 파티클 수에 비례

	for (int i = 0; i < randParticle; ++i) {
		Particle p;

		p.m = rand() / (double)RAND_MAX * 20.0;

		p.x[0] = x;
		p.x[1] = Height - y;
		p.x[2] = 0.0;

		double theta = 2 * 3.14 * (double)i / (randParticle - 1);
		double speed = rand() / (double)RAND_MAX * 10.0f;
		p.v[0] = speed * cos(theta);
		p.v[1] = speed * sin(theta);
		p.v[2] = 0.0;

		p.launchV[0] = 0.0f;
		p.launchV[1] = launchSpeed;
		p.launchV[2] = 0.0f;

		p.size = rand() / (double)RAND_MAX * 5.0;

		p.c[0] = r;
		p.c[1] = g;
		p.c[2] = b;

		p.launch = true;
		p.launchTime = time;
		p.age = p.m + p.launchTime * 1.2;

		PSystem.push_back(p);
	}
}

void Render() {
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, Width, 0.0, Height, -10.0, 10.0);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	glEnable(GL_POINT_SMOOTH);
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

	hipGraphicsGLRegisterBuffer(&cuda_pbo, gl_pbo, cudaGraphicsMapFlagsNone);

	// 4. 픽셀 버퍼를 CUDA 시스템에 매핑하고, 매핑된 GPU 주소(pDevImage)를 구함
	size_t size;
	hipGraphicsMapResources(1, &cuda_pbo, NULL);
	hipGraphicsResourceGetMappedPointer((void**)&pDevImage, &size, cuda_pbo);

	// Julia 집합 찾아 픽셀 버퍼를 채운다.
	//CreateJuliaSet();
	
	for (int i = 0; i < PSystem.size(); ++i) {
		glPointSize(PSystem[i].size);
		glBegin(GL_POINTS);
		glColor3dv(PSystem[i].c);
		glVertex3dv(PSystem[i].x);
		glEnd();
	}
	glDrawPixels(Width, Height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glFinish();

	// 5. GPU 리소스를 해제한다.
	hipGraphicsUnmapResources(1, &cuda_pbo, NULL);
	hipGraphicsUnregisterResource(cuda_pbo);

	glutSwapBuffers();
}

void Keyboard(unsigned char key, int x, int y) {
	if (key == 27) exit(1);

	if (key == '1') {
		ExtForce[0] = 100.0;
		ExtForce[1] = 0.0;
		ExtForce[2] = 0.0;
	}
	if (key == '2') {
		ExtForce[0] = -100.0;
		ExtForce[1] = 0.0;
		ExtForce[2] = 0.0;
	}
}