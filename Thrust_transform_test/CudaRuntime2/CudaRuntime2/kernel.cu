#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "..\usr\include\GL\freeglut.h"
#include <vector>
#include "..\thrust\device_vector.h"
#include "..\thrust\host_vector.h"
//#include "..\thrust\transform.h"
#include <chrono>

using namespace std;
using namespace thrust;

int Width = 800;
int Height = 600;

struct Particle {
	double m;		// 질량
	double x[3];	// 위치
	double v[3];	// 속도
	double size;	// 크기
	double c[3];	// 색상
};

host_vector<Particle> HSystem;

void Render();
void Reshape(int w, int h);
void Mouse(int button, int state, int x, int y);
void Timer(int id);

void iter(double dt, device_vector<Particle> DSystem);

struct changeValue {
	__host__ __device__ Particle operator()(Particle p) {
		double Gravity[3] = { 0.0, -9.8, 0.0 };
		p.x[1] = p.x[1] + 0.1 * p.v[1];
		p.x[2] = p.x[2] + 0.1 * p.v[2];
		p.x[3] = p.x[3] + 0.1 * p.v[3];

		p.v[0] = p.v[0] + 0.1 * Gravity[0];
		p.v[1] = p.v[1] + 0.1 * Gravity[1];
		p.v[2] = p.v[2] + 0.1 * Gravity[2];

		return p;
	}
};

int main(int argc, char** argv) {
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);

	glutInitWindowSize(Width, Height);
	glutCreateWindow("Particle System");

	glutDisplayFunc(Render);
	glutReshapeFunc(Reshape);
	glutMouseFunc(Mouse);
	glutTimerFunc(10, Timer, 0);

	hipSetDevice(0);
	//hipMalloc((void**)&DevImage, sizeof(unsigned char) * Dim * Dim * 3);*/

	glutMainLoop();
	return 0;
}

void Reshape(int w, int h) {
	glViewport(0, 0, w, h);
}

// GPU 연산 필요한 부분
void iter(double dt, device_vector<Particle> DSystem) {
	device_vector<Particle>::iterator it = DSystem.begin();
	printf("ITER\n");
	int i = 0;
	while (it != DSystem.end()) {
		//Particle temp = *it;
		changeValue cv;
		thrust::transform(DSystem[i], DSystem[i+1], DSystem[i], cv);
		//transform(DSystem.begin(), DSystem.end(), DSystem.begin(), )
		
		/*if (it[0].operator Particle().x[1] < 0.0) {
			printf("????????????????????????????????????????????????????????????????\n");
			it = DSystem.erase(it);
			continue;
		}*/
		++it;
		i++;
	}
	
}

void Timer(int id) {
	device_vector<Particle> DSystem = HSystem;
	//DSystem = HSystem;
	double dt = 0.1;

	Particle* ptr = thrust::raw_pointer_cast(DSystem.data());

	dim3 gridsize((HSystem.size() - 1) / 32 + 1, (HSystem.size() - 1) / 32 + 1, 1);
	dim3 blocksize(32, 32, 1);

	//GPUiter<<<gridsize, blocksize>>>(dt, ptr)

	printf("TIMER\n");
	iter(dt, DSystem);
	hipDeviceSynchronize();
	//HSystem = DSystem;
	thrust::copy(DSystem.begin(), DSystem.end(), HSystem.begin());

	glutPostRedisplay();
	glutTimerFunc(10, Timer, 0);
}

void Mouse(int button, int state, int x, int y) {
	printf("MOUSE\n");
	int NumParticle = 1000;
	for (int i = 0; i < NumParticle; ++i) {
		Particle p;

		p.m = rand() / (double)RAND_MAX * 10.0;

		p.x[0] = x;
		p.x[1] = Height - y;
		p.x[2] = 0.0;

		double theta = 2 * 3.14 * (double)i / (NumParticle - 1);
		double speed = rand() / (double)RAND_MAX * 10.0f;
		p.v[0] = speed * cos(theta);
		p.v[1] = speed * sin(theta);
		p.v[2] = 0.0;

		p.size = rand() / (double)RAND_MAX * 5.0;
		p.c[0] = rand() / (double)RAND_MAX;
		p.c[1] = rand() / (double)RAND_MAX;
		p.c[2] = rand() / (double)RAND_MAX;

		HSystem.push_back(p);
	}
}

void createParticle() {


	for (int i = 0; i < HSystem.size(); ++i) {
		printf("RENDER\n");
		glPointSize(HSystem[i].size);
		glBegin(GL_POINTS);
		glColor3dv(HSystem[i].c);
		glVertex3dv(HSystem[i].x);
		glEnd();
	}
}

void Render() {
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, Width, 0.0, Height, -10.0, 10.0);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	glEnable(GL_POINT_SMOOTH);
	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

	//printf("RENDER\n");

	createParticle();
	//glDrawPixels(Dim, Dim, GL_RGB, GL_UNSIGNED_BYTE, Image);

	glutSwapBuffers();
}